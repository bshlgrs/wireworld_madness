#include "hip/hip_runtime.h"
/**
 * A CUDA 2.0 Implementation of the Game of Life.
 *
 * Copyright (C) 2015 Alejandro Segovia
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *        
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *            
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 **/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>

#define BLOCK_SIDE 16

typedef unsigned char ubyte;

__device__ ubyte getat(ubyte* pboard, int nrows, int ncols, int x, int y)
{
  if (x >= 0 && x < ncols && y >= 0 && y < nrows)
    return pboard[x * ncols + y];
  return 0x0;
}

__device__ int numneighbors(int x, int y, ubyte* pboard, int nrows, int ncols)
{
  int num = 0;

  num += (getat(pboard, nrows, ncols, x-1, y));

  num += (getat(pboard, nrows, ncols, x+1, y));
  
  num += (getat(pboard, nrows, ncols, x, y-1));
  
  num += (getat(pboard, nrows, ncols, x, y+1));
  
  num += (getat(pboard, nrows, ncols, x-1, y-1));
  
  num += (getat(pboard, nrows, ncols, x-1, y+1));
  
  num += (getat(pboard, nrows, ncols, x+1, y-1));
  
  num += (getat(pboard, nrows, ncols, x+1, y+1));
  
  return num;
}

__global__ void simstep(int nrows, int ncols, ubyte* pCurrBoard, ubyte* pNewBoard)
{
  int x = blockIdx.x * BLOCK_SIDE + threadIdx.x;
  int y = blockIdx.y * BLOCK_SIDE + threadIdx.y;

  int indx = x * ncols + y;

  pNewBoard[indx] = pCurrBoard[indx];

  int neighbors = numneighbors(x, y, pCurrBoard, nrows, ncols);

  // Apply game rules:
  // "Any live cell with fewer than two live neighbours dies, 
  // as if caused by under-population." [WIK11]
  if (neighbors < 2)
    pNewBoard[indx] = 0x0;

  // "Any live cell with two or three live neighbours lives on 
  // to the next generation." [WIK11]
  // (do nothing)

  // "Any live cell with more than three live neighbours dies, 
  // as if by overcrowding." [WIK11]
  if (neighbors > 3)
    pNewBoard[indx] = 0x0;

  // "Any dead cell with exactly three live neighbours becomes 
  // a live cell, as if by reproduction." [WIK11]
  if (neighbors == 3 && !pCurrBoard[indx])
    pNewBoard[indx] = 0x1;
}

void randomizeBoard(ubyte* pboard, int nrows, int ncols, float probability)
{
  for (int x = 0; x < ncols; x++)
  {
    for (int y = 0; y < nrows; y++)
    {
      float rnd = rand() / (float)RAND_MAX;
      pboard[x * ncols + y] = (rnd >= probability)? 0x1 : 0x0;
    }
  }
}

void printBoard(const char* msg, ubyte* pboard, int nrows, int ncols)
{
  printf("%s\n", msg);

  for (int x = 0; x < ncols; x++)
  {
    for (int y = 0; y < nrows; y++)
    {
      printf("%c ", pboard[x * ncols + y]? 'o' : ' ');
    }
    printf("\n");
  }

}

int main(int argc, char* argv[])
{
  int boardW = 64;
  int boardH = 64;

  int ngenerations = 1000000;
  if (argc > 1)
  {
    ngenerations = atoi(argv[1]);
  }

  printf("Running %d generations\n", ngenerations);

  srand(time(0));

  ubyte* pboard = (ubyte *)malloc(boardW * boardH * sizeof(ubyte));
  randomizeBoard(pboard, boardH, boardW, 0.7f);
  printBoard("Initial Board:", pboard, boardH, boardW);

  ubyte* pDevBoard0;
  hipMalloc((void **)&pDevBoard0, boardW * boardH * sizeof(ubyte));
  hipMemcpy(pDevBoard0, pboard, boardH * boardW * sizeof(ubyte), hipMemcpyHostToDevice);

  ubyte* pDevBoard1;
  hipMalloc((void **)&pDevBoard1, boardW * boardH * sizeof(ubyte));
  hipMemset(pDevBoard1, 0x0, boardH * boardW * sizeof(ubyte));

  dim3 blocksize(BLOCK_SIDE, BLOCK_SIDE);
  dim3 gridsize(boardW / BLOCK_SIDE, boardH / BLOCK_SIDE);

  struct timeval ti;
  gettimeofday(&ti, NULL);

  ubyte* pcurr;
  ubyte* pnext;
  for (int gen = 0; gen < ngenerations; gen++)
  {
    if ((gen % 2) == 0)
    {
      pcurr = pDevBoard0;
      pnext = pDevBoard1;
    }
    else
    {
      pcurr = pDevBoard1;
      pnext = pDevBoard0;
    }

    simstep<<<gridsize, blocksize>>>(boardH, boardW, pcurr, pnext);

#ifdef PRINT_BOARDS
    hipMemcpy(pboard, pnext, boardH * boardW * sizeof(ubyte), hipMemcpyDeviceToHost);
    for (int i = 0; i < 24; i++) printf("\n");
    printBoard(" ", pboard, boardH, boardW);
    //usleep(250000);
#endif

  }

  struct timeval tf;
  gettimeofday(&tf, NULL);
  double t = ((tf.tv_sec - ti.tv_sec) * 1000.0) + ((tf.tv_usec - ti.tv_usec) / 1000.0);


  hipMemcpy(pboard, pcurr, boardW * boardH * sizeof(ubyte), hipMemcpyDeviceToHost);

  printBoard("Resulting Board:", pboard, boardH, boardW);

  hipFree(pDevBoard0);
  hipFree(pDevBoard1);
  free(pboard);

  printf("%d generations in %f milliseconds\n", ngenerations, t);

  return 0;
}
