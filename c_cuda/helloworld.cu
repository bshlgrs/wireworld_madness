
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 512

__global__ void add(int *a, int *b, int *c) {
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main(void) {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  a = (int *)malloc(size);
  // random_ints(a, N);
  b = (int *)malloc(size);
  // random_ints(b, N);
  c = (int *)malloc(size);

  a[0] = 10;
  b[0] = 7;

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);


  add<<<N,1>>>(d_a, d_b, d_c);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  printf("Hello World! %d %d %d, %d %d %d\n", a[0], b[0], c[0], a[1], b[1], c[1]);
  return 0;
}